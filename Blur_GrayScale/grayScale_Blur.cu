#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

#define NUM_CHANNELS 3

__constant__ int BLUR_SIZE = 25;

/*
 * Convertir a escala de grises
*/
__global__
void colorToGreyscaleConversion(unsigned char* Pout, const unsigned char* Pin, int width, int height){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Verifica si las variables fila y columna de los subprocesos están dentro de los límites de la imagen
    if(col < width && row < height) { // CUBRE TODA LA IMAGEN
        int grey_offset = row * width + col; 
        int rgb_offset = grey_offset * NUM_CHANNELS;

        unsigned char r = Pin[rgb_offset + 0]; // rojo
        unsigned char g = Pin[rgb_offset + 1]; // verde
        unsigned char b = Pin[rgb_offset + 2]; // azul

        // Realiza la conversión de rgb a gray promediando los tres canales 
        Pout[grey_offset] = (unsigned char)(0.21f * r + 0.71f * g + 0.07f * b);
    }
}

/*
 * Difuminar
*/
__global__
void blurKernel(unsigned char* Pout, const unsigned char* Pin, int width, int height){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Verifica si las variables fila y columna de los subprocesos están dentro de los límites de la imagen
    if(col < width && row < height) {
        int pixVal = 0;
        int pixels = 0;

        // Calcula la media de los píxeles vecinos
        // Los dos bucles for iteran a través del cuadro que define el valor BLUR_SIZE
        for(int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE + 1; ++blurrow){
            for(int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE + 1; ++blurcol){
                int currow = row + blurrow;
                int curcol = col + blurcol;
            
                // Si el elemento está dentro de los límites de la imagen, acumula el valor en la variable pixVal e incrementa el contador de píxeles acumulados.
                if(currow > -1 && currow < height && curcol > -1 && curcol < width){
                    pixVal += Pin[currow * width + curcol];
                    ++pixels; // Cuenta el número de valores de píxeles que se han añadido
                }
            }
        }
    Pout[row * width + col] = (unsigned char)(pixVal / pixels);
    }
}

int main(int argc, char* argv[]){
  // Read the image
  int width, height, channels;
  unsigned char* h_Pin = stbi_load(argv[1], &width, &height, &channels, 0); 
  cout << "Imported image " << argv[1] << " (" << width << " x " << height << ") with " << channels << " channels" << endl;

  // Asigna memoria en el host para la imagen de salida
    int size = width*height;
    unsigned char* h_Pout = new unsigned char[size];

    // Asigna memoria en el device para la imagen de entrada
    unsigned char* d_Pin;
    hipMalloc((void**)&d_Pin, size*3);
    hipMemcpy((void*)d_Pin, (void*)h_Pin, size*3, hipMemcpyHostToDevice);

    // Asigna memoria para escala de grises
    unsigned char* d_Pbw;
    hipMalloc((void**)&d_Pbw, size);

    // Asigna memoria para Blur    
    unsigned char* d_Pout;
    hipMalloc((void**)&d_Pout, size);

    dim3 dimGrid(ceil(width/16.0f),ceil(height/16.0f),1);
    dim3 dimBlock(16,16,1);
    cout << "Launching a (" << dimGrid.x << " x " << dimGrid.y << " x " << dimGrid.z << ") grid." << endl;
    cout << "Total number of threads: " << dimGrid.x*dimGrid.y*dimGrid.z*16*16 << endl;
    cout << "Number of pixels: " << width*height << endl;

    // ESCALA DE GRISES
    colorToGreyscaleConversion<<<dimGrid,dimBlock>>>(d_Pbw, d_Pin, width, height);
    hipMemcpy((void*)h_Pout, (void*)d_Pbw, size, hipMemcpyDeviceToHost);
    stbi_write_bmp("grayScale.png", width, height, 1, (void*)h_Pout);


    // BLUR
    blurKernel<<<dimGrid,dimBlock>>>(d_Pout, d_Pbw, width, height);
    hipMemcpy((void*)h_Pout, (void*)d_Pout, size, hipMemcpyDeviceToHost);
    stbi_write_bmp("Blur.png", width, height, 1, (void*)h_Pout);

    hipFree(d_Pin);
    hipFree(d_Pbw);
    hipFree(d_Pout);
 
    cout << "Closing..." << endl;

    return 0;
}